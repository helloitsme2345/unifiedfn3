#include "hip/hip_runtime.h"
//
// File: InformationFilterUpdate.cu
//
// GPU Coder version                    : 1.5
// CUDA/C/C++ source code generated on  : 07-Sep-2020 12:29:29
//

// Include Files
#include "InformationFilterUpdate.h"
#include "MWCudaDimUtility.hpp"
#include <cmath>

// Function Declarations
static __global__ void InformationFilterUpdate_kernel1(double Rw[16]);
static __global__ void InformationFilterUpdate_kernel2(const double Rw[4],
  double b_Rw[16]);
static __global__ void InformationFilterUpdate_kernel3(double Rw_inv[16]);
static __global__ void InformationFilterUpdate_kernel4(signed char ipiv[4]);
static __global__ void InformationFilterUpdate_kernel5(signed char p[4]);
static __global__ void InformationFilterUpdate_kernel6(double Fk_inv[64]);
static __global__ void InformationFilterUpdate_kernel7(const double T, double x
  [64]);
static __global__ void InformationFilterUpdate_kernel8(const signed char iv[8],
  const signed char iv1[8], const signed char iv2[8], signed char ipiv[8],
  double x[64]);
static __global__ void InformationFilterUpdate_kernel9(signed char p[8]);
static __global__ void ab_InformationFilterUpdate_kern(const signed char iv3[8],
  const signed char iv4[8], const signed char iv5[8], double H_linear[104]);
static __global__ void b_InformationFilterUpdate_kerne(const double T, double
  Gk[32]);
static __global__ void bb_InformationFilterUpdate_kern(const double ih[8], const
  double H_linear[104], double hk[13]);
static __global__ void c_InformationFilterUpdate_kerne(const double a[64], const
  double Fk_inv[64], const int i, double x[64]);
static __global__ void cb_InformationFilterUpdate_kern(const double smax, const
  double delta, const double L_imuToRear, const double ih[8], double H_linear
  [104], double hk[13]);
static __global__ void d_InformationFilterUpdate_kerne(const double Fk_inv[64],
  const double x[64], const int i, double Ih[64]);
static __global__ void db_InformationFilterUpdate_kern(const double H_linear[104],
  double A[104]);
static __global__ void e_InformationFilterUpdate_kerne(double Rw[16]);
static __global__ void eb_InformationFilterUpdate_kern(const double ih[8], const
  double H_linear[104], const double hk[13], const double y_meas[13], double
  b_y_meas[13]);
static __global__ void f_InformationFilterUpdate_kerne(const double Ih[64],
  const double Gk[32], double b_Gk[32]);
static __global__ void fb_InformationFilterUpdate_kern(const double y_meas[13],
  const double C[104], const double ik[8], double op[8]);
static __global__ void g_InformationFilterUpdate_kerne(const double Gk[32],
  const double b_Gk[32], const double Rw_inv[16], double x[16]);
static __global__ void h_InformationFilterUpdate_kerne(signed char ipiv[4]);
static __global__ void i_InformationFilterUpdate_kerne(signed char p[4]);
static __global__ void j_InformationFilterUpdate_kerne(const double Gk[32],
  const double Ih[64], double b_Ih[32]);
static __global__ void k_InformationFilterUpdate_kerne(const double Rw[16],
  const double Ih[32], double b_Ih[32]);
static __global__ void l_InformationFilterUpdate_kerne(const double Gk[32],
  const double Ih[32], double calcEq[64]);
static __global__ void m_InformationFilterUpdate_kerne(double ih[8]);
static __global__ void n_InformationFilterUpdate_kerne(const double ih[8], const
  double a[64], double ik[8]);
static __global__ void o_InformationFilterUpdate_kerne(const double ik[8], const
  double Fk_inv[64], double ih[8]);
static __global__ void p_InformationFilterUpdate_kerne(const double calcEq[64],
  const double ih[8], double ik[8]);
static __global__ void q_InformationFilterUpdate_kerne(double Fk_inv[64]);
static __global__ void r_InformationFilterUpdate_kerne(const double calcEq[64],
  const double Ih[64], double x[64]);
static __global__ void s_InformationFilterUpdate_kerne(signed char ipiv[8]);
static __global__ void t_InformationFilterUpdate_kerne(signed char p[8]);
static __global__ void u_InformationFilterUpdate_kerne(const double ik[8], const
  double Fk_inv[64], double ih[8]);
static __global__ void v_InformationFilterUpdate_kerne(const double
  B_usedMeas_vec[13], const double Re[13], double Re_inv[13]);
static __global__ void w_InformationFilterUpdate_kerne(double Re_inv[169]);
static __global__ void x_InformationFilterUpdate_kerne(const double Re_inv[13],
  double b_Re_inv[169]);
static __global__ void y_InformationFilterUpdate_kerne(double H_linear[104]);

// Function Definitions

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Rw[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel1
  (double Rw[16])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 16) {
    // 4*1
    // 4*4
    // ---- Prediction step -------------------------
    Rw[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Rw[4]
//                double b_Rw[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel2(
  const double Rw[4], double b_Rw[16])
{
  unsigned int threadId;
  int j;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  j = static_cast<int>(threadId);
  if (j < 4) {
    b_Rw[j + (j << 2)] = Rw[j];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Rw_inv[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel3
  (double Rw_inv[16])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 16) {
    Rw_inv[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char ipiv[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel4
  (signed char ipiv[4])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    ipiv[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel5
  (signed char p[4])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    p[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Fk_inv[64]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void InformationFilterUpdate_kernel6
  (double Fk_inv[64])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 64) {
    //  System matrix
    // 4*4
    Fk_inv[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double T
//                double x[64]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel7(
  const double T, double x[64])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    x[52] = 0.0;
    x[60] = 0.0;
    x[6] = 0.0;
    x[14] = 0.0;
    x[22] = 0.0;
    x[30] = T;
    x[38] = 0.0;
    x[46] = 0.0;
    x[54] = 1.0;
    x[62] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char iv[8]
//                const signed char iv1[8]
//                const signed char iv2[8]
//                signed char ipiv[8]
//                double x[64]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel8(
  const signed char iv[8], const signed char iv1[8], const signed char iv2[8],
  signed char ipiv[8], double x[64])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    x[(i << 3) + 2] = static_cast<double>(iv2[i]);
    x[(i << 3) + 5] = static_cast<double>(iv1[i]);
    x[(i << 3) + 7] = static_cast<double>(iv[i]);
    ipiv[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void InformationFilterUpdate_kernel9
  (signed char p[8])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    p[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const signed char iv3[8]
//                const signed char iv4[8]
//                const signed char iv5[8]
//                double H_linear[104]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void ab_InformationFilterUpdate_kern(
  const signed char iv3[8], const signed char iv4[8], const signed char iv5[8],
  double H_linear[104])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    H_linear[13 * i] = static_cast<double>(iv5[i]);
    H_linear[13 * i + 2] = static_cast<double>(iv4[i]);
    H_linear[13 * i + 3] = static_cast<double>(iv3[i]);
    H_linear[13 * i + 4] = 0.0;
    H_linear[13 * i + 5] = 0.0;
    H_linear[13 * i + 10] = 0.0;
    H_linear[13 * i + 11] = static_cast<double>(iv3[i]);
    H_linear[13 * i + 12] = static_cast<double>(iv3[i]);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double T
//                double Gk[32]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void b_InformationFilterUpdate_kerne(
  const double T, double Gk[32])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    Gk[21] = T;
    Gk[29] = 0.0;
    Gk[6] = 0.0;
    Gk[14] = T * T / 2.0;
    Gk[22] = 0.0;
    Gk[30] = T * T / 2.0;
    Gk[7] = 0.0;
    Gk[15] = T;
    Gk[23] = 0.0;
    Gk[31] = T;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ih[8]
//                const double H_linear[104]
//                double hk[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void bb_InformationFilterUpdate_kern(
  const double ih[8], const double H_linear[104], double hk[13])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 13) {
    //  Nonlinear parts
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += H_linear[i + 13 * i4] * ih[i4];
    }

    hk[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double a[64]
//                const double Fk_inv[64]
//                const int i
//                double x[64]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void c_InformationFilterUpdate_kerne(
  const double a[64], const double Fk_inv[64], const int i, double x[64])
{
  unsigned int threadId;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId);
  if (i5 < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += Fk_inv[i4 + (i << 3)] * a[i4 + (i5 << 3)];
    }

    x[i + (i5 << 3)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double smax
//                const double delta
//                const double L_imuToRear
//                const double ih[8]
//                double H_linear[104]
//                double hk[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void cb_InformationFilterUpdate_kern(
  const double smax, const double delta, const double L_imuToRear, const double
  ih[8], double H_linear[104], double hk[13])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    hk[0] -= ih[2] * ih[2] * L_imuToRear;
    hk[1] += ih[0] * ih[2];
    hk[10] = ih[0] * cos(delta) + smax * ih[2] * sin(delta);

    // updates the Hk 13*4matrix HK(5,1)=Hk matrix's 5th row 1st element
    // being updated with cos(delta)value . Likewise for all.
    H_linear[26] = -2.0 * ih[2] * L_imuToRear;
    H_linear[1] = ih[2];
    H_linear[27] = ih[0];
    H_linear[4] = cos(delta);
    H_linear[10] = cos(delta);
    H_linear[36] = smax * sin(delta);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Fk_inv[64]
//                const double x[64]
//                const int i
//                double Ih[64]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void d_InformationFilterUpdate_kerne(
  const double Fk_inv[64], const double x[64], const int i, double Ih[64])
{
  unsigned int threadId;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId);
  if (i5 < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += x[i + (i4 << 3)] * Fk_inv[i4 + (i5 << 3)];
    }

    Ih[i + (i5 << 3)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double H_linear[104]
//                double A[104]
// Return Type  : void
//
static __global__ __launch_bounds__(128, 1) void db_InformationFilterUpdate_kern
  (const double H_linear[104], double A[104])
{
  unsigned int threadId;
  int i;
  int i4;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i4 = static_cast<int>(threadId % 8U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i4)) / 8U);
  if (i < 13) {
    A[i4 + (i << 3)] = H_linear[i + 13 * i4];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Rw[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void e_InformationFilterUpdate_kerne
  (double Rw[16])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 16) {
    // 4*4
    Rw[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ih[8]
//                const double H_linear[104]
//                const double hk[13]
//                const double y_meas[13]
//                double b_y_meas[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void eb_InformationFilterUpdate_kern(
  const double ih[8], const double H_linear[104], const double hk[13], const
  double y_meas[13], double b_y_meas[13])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 13) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += H_linear[i + 13 * i4] * ih[i4];
    }

    b_y_meas[i] = (y_meas[i] - hk[i]) + d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Ih[64]
//                const double Gk[32]
//                double b_Gk[32]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void f_InformationFilterUpdate_kerne(
  const double Ih[64], const double Gk[32], double b_Gk[32])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 8U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 8U);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += Gk[i4 + (i << 3)] * Ih[i4 + (i5 << 3)];
    }

    b_Gk[i + (i5 << 2)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double y_meas[13]
//                const double C[104]
//                const double ik[8]
//                double op[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void fb_InformationFilterUpdate_kern(
  const double y_meas[13], const double C[104], const double ik[8], double op[8])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 13; i4++) {
      d += C[i + (i4 << 3)] * y_meas[i4];
    }

    op[i] = ik[i] + d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Gk[32]
//                const double b_Gk[32]
//                const double Rw_inv[16]
//                double x[16]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void g_InformationFilterUpdate_kerne(
  const double Gk[32], const double b_Gk[32], const double Rw_inv[16], double x
  [16])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 4U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 4U);
  if (i < 4) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += b_Gk[i + (i4 << 2)] * Gk[i4 + (i5 << 3)];
    }

    x[i + (i5 << 2)] = d + Rw_inv[i + (i5 << 2)];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char ipiv[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void h_InformationFilterUpdate_kerne
  (signed char ipiv[4])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    ipiv[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[4]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void i_InformationFilterUpdate_kerne
  (signed char p[4])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 4) {
    p[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Gk[32]
//                const double Ih[64]
//                double b_Ih[32]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void j_InformationFilterUpdate_kerne(
  const double Gk[32], const double Ih[64], double b_Ih[32])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 4U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 4U);
  if (i < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += Ih[i + (i4 << 3)] * Gk[i4 + (i5 << 3)];
    }

    b_Ih[i + (i5 << 3)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Rw[16]
//                const double Ih[32]
//                double b_Ih[32]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void k_InformationFilterUpdate_kerne(
  const double Rw[16], const double Ih[32], double b_Ih[32])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 4U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 4U);
  if (i < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += Ih[i + (i4 << 3)] * Rw[i4 + (i5 << 2)];
    }

    b_Ih[i + (i5 << 3)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Gk[32]
//                const double Ih[32]
//                double calcEq[64]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void l_InformationFilterUpdate_kerne(
  const double Gk[32], const double Ih[32], double calcEq[64])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 8U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 8U);
  if (i < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 4; i4++) {
      d += Ih[i + (i4 << 3)] * Gk[i5 + (i4 << 3)];
    }

    calcEq[i + (i5 << 3)] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double ih[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void m_InformationFilterUpdate_kerne
  (double ih[8])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    ih[5] = 0.0;
    ih[6] = 0.0;
    ih[7] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ih[8]
//                const double a[64]
//                double ik[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void n_InformationFilterUpdate_kerne(
  const double ih[8], const double a[64], double ik[8])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += a[i + (i4 << 3)] * ih[i4];
    }

    ik[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ik[8]
//                const double Fk_inv[64]
//                double ih[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void o_InformationFilterUpdate_kerne(
  const double ik[8], const double Fk_inv[64], double ih[8])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += Fk_inv[i4 + (i << 3)] * ik[i4];
    }

    ih[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double calcEq[64]
//                const double ih[8]
//                double ik[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void p_InformationFilterUpdate_kerne(
  const double calcEq[64], const double ih[8], double ik[8])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    // 4*1
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += calcEq[i + (i4 << 3)] * ih[i4];
    }

    ik[i] = ih[i] - d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Fk_inv[64]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void q_InformationFilterUpdate_kerne
  (double Fk_inv[64])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 64) {
    // 4*1
    Fk_inv[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double calcEq[64]
//                const double Ih[64]
//                double x[64]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void r_InformationFilterUpdate_kerne(
  const double calcEq[64], const double Ih[64], double x[64])
{
  unsigned int threadId;
  int i;
  double d;
  int i5;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i5 = static_cast<int>(threadId % 8U);
  i = static_cast<int>((threadId - static_cast<unsigned int>(i5)) / 8U);
  if (i < 8) {
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += calcEq[i + (i4 << 3)] * Ih[i4 + (i5 << 3)];
    }

    x[i + (i5 << 3)] = Ih[i + (i5 << 3)] - d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char ipiv[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void s_InformationFilterUpdate_kerne
  (signed char ipiv[8])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    ipiv[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                signed char p[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void t_InformationFilterUpdate_kerne
  (signed char p[8])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    p[i] = static_cast<signed char>(i + 1);
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double ik[8]
//                const double Fk_inv[64]
//                double ih[8]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void u_InformationFilterUpdate_kerne(
  const double ik[8], const double Fk_inv[64], double ih[8])
{
  unsigned int threadId;
  double d;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 8) {
    // 4*4
    d = 0.0;
    for (int i4 = 0; i4 < 8; i4++) {
      d += Fk_inv[i + (i4 << 3)] * ik[i4];
    }

    ih[i] = d;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double B_usedMeas_vec[13]
//                const double Re[13]
//                double Re_inv[13]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void v_InformationFilterUpdate_kerne(
  const double B_usedMeas_vec[13], const double Re[13], double Re_inv[13])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 13) {
    // 1*1
    Re_inv[i] = 1.0 / Re[i] * B_usedMeas_vec[i];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Re_inv[169]
// Return Type  : void
//
static __global__ __launch_bounds__(192, 1) void w_InformationFilterUpdate_kerne
  (double Re_inv[169])
{
  unsigned int threadId;
  int i;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  i = static_cast<int>(threadId);
  if (i < 169) {
    // 1*13
    Re_inv[i] = 0.0;
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Re_inv[13]
//                double b_Re_inv[169]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void x_InformationFilterUpdate_kerne(
  const double Re_inv[13], double b_Re_inv[169])
{
  unsigned int threadId;
  int j;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  j = static_cast<int>(threadId);
  if (j < 13) {
    b_Re_inv[j + 13 * j] = Re_inv[j];
  }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double H_linear[104]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void y_InformationFilterUpdate_kerne
  (double H_linear[104])
{
  unsigned int threadId;
  int tmpIdx;
  threadId = static_cast<unsigned int>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int>(threadId);
  if (tmpIdx < 1) {
    H_linear[100] = 0.0;
  }
}

//
// Arguments    : const double y_meas[13]
//                const double B_usedMeas_vec[13]
//                const double initialization_vec[4]
//                double delta
//                const double Rw[4]
//                const double Re[13]
//                double L_imuToRear
//                double L_geometricWheelbase
//                const double L_trackWidth[5]
//                const double L_axlePos[5]
//                double T
//                double op[8]
// Return Type  : void
//
void InformationFilterUpdate(const double y_meas[13], const double
  B_usedMeas_vec[13], const double initialization_vec[4], double delta, const
  double Rw[4], const double Re[13], double L_imuToRear, double
  L_geometricWheelbase, const double L_trackWidth[5], const double L_axlePos[5],
  double T, double op[8])
{
  int j;
  int c;
  int ar;
  int iy;
  int ia;
  int ix;
  double smax;
  signed char i1;
  double s;
  int i2;
  int b_j;
  int jy;
  int i;
  int kAcol;
  int i3;
  static const signed char iv[8] = { 0, 0, 0, 0, 0, 0, 0, 1 };

  static const signed char iv1[8] = { 0, 0, 0, 0, 0, 1, 0, 0 };

  static const signed char iv2[8] = { 0, 0, 1, 0, 0, 0, 0, 0 };

  static const double a[64] = { 100.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
    33.333333333333336, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 10000.0, 0.0,
    0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 500.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
    0.0, 100.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 33.333333333333336, 0.0,
    0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 10000.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
    0.0, 0.0, 500.0 };

  static const signed char iv3[8] = { 1, 0, 0, 0, 1, 0, 0, 0 };

  static const signed char iv4[8] = { 0, 0, 1, 0, 0, 0, 1, 0 };

  static const signed char iv5[8] = { 0, 1, 0, 0, 0, 1, 0, 0 };

  double (*gpu_Rw)[16];
  double (*b_gpu_Rw)[4];
  double (*gpu_Rw_inv)[16];
  signed char (*gpu_ipiv)[4];
  signed char (*gpu_p)[4];
  double (*gpu_Fk_inv)[64];
  double (*gpu_x)[64];
  signed char (*gpu_iv)[8];
  signed char (*gpu_iv1)[8];
  signed char (*gpu_iv2)[8];
  signed char (*b_gpu_ipiv)[8];
  signed char (*b_gpu_p)[8];
  double (*gpu_Gk)[32];
  double (*gpu_a)[64];
  double (*gpu_Ih)[64];
  double (*b_gpu_Gk)[32];
  double (*b_gpu_x)[16];
  double (*b_gpu_Ih)[32];
  double (*c_gpu_Ih)[32];
  double (*gpu_calcEq)[64];
  double (*gpu_ih)[8];
  double (*gpu_ik)[8];
  double (*gpu_B_usedMeas_vec)[13];
  double (*gpu_Re)[13];
  double (*gpu_Re_inv)[13];
  double (*b_gpu_Re_inv)[169];
  double (*gpu_H_linear)[104];
  signed char (*gpu_iv3)[8];
  signed char (*gpu_iv4)[8];
  signed char (*gpu_iv5)[8];
  double (*gpu_hk)[13];
  double (*gpu_A)[104];
  double (*gpu_y_meas)[13];
  double (*b_gpu_y_meas)[13];
  double (*gpu_C)[104];
  double (*gpu_op)[8];
  bool syncIsDirty;
  hipMallocManaged(&gpu_C, 832UL);
  hipMallocManaged(&b_gpu_y_meas, 104UL);
  hipMallocManaged(&gpu_A, 832UL);
  hipMallocManaged(&gpu_hk, 104UL);
  hipMallocManaged(&gpu_H_linear, 832UL);
  hipMallocManaged(&b_gpu_Re_inv, 1352UL);
  hipMallocManaged(&gpu_Re_inv, 104UL);
  hipMallocManaged(&gpu_ik, 64UL);
  hipMallocManaged(&gpu_ih, 64UL);
  hipMallocManaged(&gpu_calcEq, 512UL);
  hipMallocManaged(&c_gpu_Ih, 256UL);
  hipMallocManaged(&b_gpu_Ih, 256UL);
  hipMallocManaged(&b_gpu_x, 128UL);
  hipMallocManaged(&b_gpu_Gk, 256UL);
  hipMallocManaged(&gpu_Ih, 512UL);
  hipMallocManaged(&gpu_Gk, 256UL);
  hipMallocManaged(&b_gpu_p, 8UL);
  hipMallocManaged(&b_gpu_ipiv, 8UL);
  hipMallocManaged(&gpu_x, 512UL);
  hipMallocManaged(&gpu_Fk_inv, 512UL);
  hipMallocManaged(&gpu_p, 4UL);
  hipMallocManaged(&gpu_ipiv, 4UL);
  hipMallocManaged(&gpu_Rw_inv, 128UL);
  hipMallocManaged(&gpu_Rw, 128UL);
  hipMallocManaged(&gpu_op, 64UL);
  hipMallocManaged(&b_gpu_Rw, 32UL);
  hipMallocManaged(&gpu_iv, 8UL);
  hipMallocManaged(&gpu_iv1, 8UL);
  hipMallocManaged(&gpu_iv2, 8UL);
  hipMallocManaged(&gpu_a, 512UL);
  hipMallocManaged(&gpu_B_usedMeas_vec, 104UL);
  hipMallocManaged(&gpu_Re, 104UL);
  hipMallocManaged(&gpu_iv3, 8UL);
  hipMallocManaged(&gpu_iv4, 8UL);
  hipMallocManaged(&gpu_iv5, 8UL);
  hipMallocManaged(&gpu_y_meas, 104UL);
  hipMemcpy(gpu_y_meas, (void *)&y_meas[0], 104UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv5, (void *)&iv5[0], 8UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv4, (void *)&iv4[0], 8UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv3, (void *)&iv3[0], 8UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_Re, (void *)&Re[0], 104UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_B_usedMeas_vec, (void *)&B_usedMeas_vec[0], 104UL,
             hipMemcpyHostToDevice);
  hipMemcpy(gpu_a, (void *)&a[0], 512UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv2, (void *)&iv2[0], 8UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv1, (void *)&iv1[0], 8UL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_iv, (void *)&iv[0], 8UL, hipMemcpyHostToDevice);
  hipMemcpy(b_gpu_Rw, (void *)&Rw[0], 32UL, hipMemcpyHostToDevice);

  // 4*1
  // 4*4
  // ---- Prediction step -------------------------
  InformationFilterUpdate_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Rw);
  InformationFilterUpdate_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_Rw, *gpu_Rw);
  InformationFilterUpdate_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Rw_inv);
  InformationFilterUpdate_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ipiv);
  syncIsDirty = true;
  for (j = 0; j < 3; j++) {
    c = j * 5;
    ar = 2 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*gpu_Rw)[c]);
    for (ia = 0; ia <= ar; ia++) {
      ix++;
      s = std::abs((*gpu_Rw)[ix]);
      if (s > smax) {
        iy = ia + 1;
        smax = s;
      }
    }

    if ((*gpu_Rw)[c + iy] != 0.0) {
      if (iy != 0) {
        (*gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        ar = iy;
        for (ia = 0; ia < 4; ia++) {
          ix = j + ia * 4;
          iy = ar + ia * 4;
          smax = (*gpu_Rw)[ix];
          (*gpu_Rw)[ix] = (*gpu_Rw)[iy];
          (*gpu_Rw)[iy] = smax;
        }
      }

      i2 = (c - j) + 2;
      for (i = 0; i <= i2 - c; i++) {
        iy = (c + i) + 1;
        (*gpu_Rw)[iy] /= (*gpu_Rw)[c];
      }
    }

    ar = 2 - j;
    iy = c + 6;
    jy = c + 4;
    for (b_j = 0; b_j <= ar; b_j++) {
      smax = (*gpu_Rw)[jy];
      if ((*gpu_Rw)[jy] != 0.0) {
        ix = c;
        i2 = iy - 2;
        i3 = iy - j;
        for (kAcol = 0; kAcol <= i3 - i2; kAcol++) {
          i = (iy + kAcol) - 1;
          (*gpu_Rw)[i] += (*gpu_Rw)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 4;
      iy += 4;
    }
  }

  InformationFilterUpdate_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_p);
  syncIsDirty = true;
  for (ia = 0; ia < 3; ia++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*gpu_ipiv)[ia] > ia + 1) {
      iy = (*gpu_p)[(*gpu_ipiv)[ia] - 1];
      (*gpu_p)[(*gpu_ipiv)[ia] - 1] = (*gpu_p)[ia];
      (*gpu_p)[ia] = static_cast<signed char>(iy);
    }
  }

  for (ia = 0; ia < 4; ia++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    i1 = (*gpu_p)[ia];
    (*gpu_Rw_inv)[ia + (((*gpu_p)[ia] - 1) << 2)] = 1.0;
    for (j = 0; j <= 3 - ia; j++) {
      b_j = ia + j;
      if ((*gpu_Rw_inv)[b_j + ((i1 - 1) << 2)] != 0.0) {
        for (i = 0; i <= 2 - b_j; i++) {
          iy = (b_j + i) + 1;
          (*gpu_Rw_inv)[iy + ((i1 - 1) << 2)] -= (*gpu_Rw_inv)[b_j + ((i1 - 1) <<
            2)] * (*gpu_Rw)[iy + (b_j << 2)];
        }
      }
    }
  }

  for (j = 0; j < 4; j++) {
    iy = (j << 2) - 1;
    for (ia = 0; ia < 4; ia++) {
      jy = 4 - ia;
      kAcol = (3 - ia) << 2;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Rw_inv)[(iy - ia) + 4] != 0.0) {
        (*gpu_Rw_inv)[(iy - ia) + 4] /= (*gpu_Rw)[(kAcol - ia) + 3];
        for (i = 0; i <= jy - 2; i++) {
          (*gpu_Rw_inv)[(i + iy) + 1] -= (*gpu_Rw_inv)[(iy - ia) + 4] * (*gpu_Rw)
            [i + kAcol];
        }
      }
    }
  }

  //  System matrix
  // 4*4
  InformationFilterUpdate_kernel6<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*gpu_Fk_inv);
  hipDeviceSynchronize();
  (*gpu_x)[0] = 1.0;
  (*gpu_x)[8] = T;
  (*gpu_x)[16] = 0.0;
  (*gpu_x)[24] = 0.0;
  (*gpu_x)[32] = 0.0;
  (*gpu_x)[40] = 0.0;
  (*gpu_x)[48] = 0.0;
  (*gpu_x)[56] = 0.0;
  (*gpu_x)[1] = 0.0;
  (*gpu_x)[9] = 1.0;
  (*gpu_x)[17] = 0.0;
  (*gpu_x)[25] = 0.0;
  (*gpu_x)[33] = 0.0;
  (*gpu_x)[41] = 1.0;
  (*gpu_x)[49] = 0.0;
  (*gpu_x)[57] = T;
  (*gpu_x)[3] = 0.0;
  (*gpu_x)[11] = 0.0;
  (*gpu_x)[19] = 0.0;
  (*gpu_x)[27] = 1.0;
  (*gpu_x)[35] = 0.0;
  (*gpu_x)[43] = 0.0;
  (*gpu_x)[51] = 0.0;
  (*gpu_x)[59] = T;
  (*gpu_x)[4] = 0.0;
  (*gpu_x)[12] = T;
  (*gpu_x)[20] = 0.0;
  (*gpu_x)[28] = 0.0;
  (*gpu_x)[36] = 1.0;
  (*gpu_x)[44] = 0.0;
  InformationFilterUpdate_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(T,
    *gpu_x);
  InformationFilterUpdate_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_iv, *gpu_iv1, *gpu_iv2, *b_gpu_ipiv, *gpu_x);
  syncIsDirty = true;
  for (j = 0; j < 7; j++) {
    c = j * 9;
    kAcol = c - 4;
    ar = 6 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*gpu_x)[c]);
    for (ia = 0; ia <= ar; ia++) {
      ix++;
      s = std::abs((*gpu_x)[ix]);
      if (s > smax) {
        iy = ia + 1;
        smax = s;
      }
    }

    if ((*gpu_x)[c + iy] != 0.0) {
      if (iy != 0) {
        (*b_gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        ar = iy;
        for (ia = 0; ia < 8; ia++) {
          ix = j + ia * 8;
          iy = ar + ia * 8;
          smax = (*gpu_x)[ix];
          (*gpu_x)[ix] = (*gpu_x)[iy];
          (*gpu_x)[iy] = smax;
        }
      }

      i2 = (c - j) + 2;
      for (i = 0; i <= i2 - kAcol; i++) {
        iy = (c + i) + 1;
        (*gpu_x)[iy] /= (*gpu_x)[c];
      }
    }

    ar = 6 - j;
    iy = c + 10;
    jy = c + 8;
    for (b_j = 0; b_j <= ar; b_j++) {
      smax = (*gpu_x)[jy];
      if ((*gpu_x)[jy] != 0.0) {
        ix = c;
        i2 = iy - 6;
        i3 = iy - j;
        for (kAcol = 0; kAcol <= i3 - i2; kAcol++) {
          i = (iy + kAcol) - 1;
          (*gpu_x)[i] += (*gpu_x)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 8;
      iy += 8;
    }
  }

  InformationFilterUpdate_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_p);
  syncIsDirty = true;
  for (ia = 0; ia < 7; ia++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*b_gpu_ipiv)[ia] > ia + 1) {
      iy = (*b_gpu_p)[(*b_gpu_ipiv)[ia] - 1];
      (*b_gpu_p)[(*b_gpu_ipiv)[ia] - 1] = (*b_gpu_p)[ia];
      (*b_gpu_p)[ia] = static_cast<signed char>(iy);
    }
  }

  for (ia = 0; ia < 8; ia++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    i1 = (*b_gpu_p)[ia];
    (*gpu_Fk_inv)[ia + (((*b_gpu_p)[ia] - 1) << 3)] = 1.0;
    for (j = 0; j <= 7 - ia; j++) {
      b_j = ia + j;
      if ((*gpu_Fk_inv)[b_j + ((i1 - 1) << 3)] != 0.0) {
        for (i = 0; i <= 6 - b_j; i++) {
          iy = (b_j + i) + 1;
          (*gpu_Fk_inv)[iy + ((i1 - 1) << 3)] -= (*gpu_Fk_inv)[b_j + ((i1 - 1) <<
            3)] * (*gpu_x)[iy + (b_j << 3)];
        }
      }
    }
  }

  for (j = 0; j < 8; j++) {
    iy = (j << 3) - 1;
    for (ia = 0; ia < 8; ia++) {
      jy = 8 - ia;
      kAcol = (7 - ia) << 3;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Fk_inv)[(iy - ia) + 8] != 0.0) {
        (*gpu_Fk_inv)[(iy - ia) + 8] /= (*gpu_x)[(kAcol - ia) + 7];
        for (i = 0; i <= jy - 2; i++) {
          (*gpu_Fk_inv)[(i + iy) + 1] -= (*gpu_Fk_inv)[(iy - ia) + 8] * (*gpu_x)
            [i + kAcol];
        }
      }
    }
  }

  //  Noise matrix
  if (syncIsDirty) {
    hipDeviceSynchronize();
  }

  (*gpu_Gk)[0] = T * T / 2.0;
  (*gpu_Gk)[8] = 0.0;
  (*gpu_Gk)[16] = T * T / 2.0;
  (*gpu_Gk)[24] = 0.0;
  (*gpu_Gk)[1] = T;
  (*gpu_Gk)[9] = 0.0;
  (*gpu_Gk)[17] = T;
  (*gpu_Gk)[25] = 0.0;
  (*gpu_Gk)[2] = 0.0;
  (*gpu_Gk)[10] = T * T / 2.0;
  (*gpu_Gk)[18] = 0.0;
  (*gpu_Gk)[26] = T * T / 2.0;
  (*gpu_Gk)[3] = 0.0;
  (*gpu_Gk)[11] = T;
  (*gpu_Gk)[19] = 0.0;
  (*gpu_Gk)[27] = T;
  (*gpu_Gk)[4] = T * T / 2.0;
  (*gpu_Gk)[12] = 0.0;
  (*gpu_Gk)[20] = T * T / 2.0;
  (*gpu_Gk)[28] = 0.0;
  (*gpu_Gk)[5] = T;
  (*gpu_Gk)[13] = 0.0;
  b_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(T,
    *gpu_Gk);

  //  Prediction step alternative 2. Gives easier matrix to invert
  for (i = 0; i < 8; i++) {
    c_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_a, *gpu_Fk_inv, i, *gpu_x);
    d_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*gpu_Fk_inv, *gpu_x, i, *gpu_Ih);
  }

  // 4*4
  e_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Rw);
  f_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Ih, *gpu_Gk, *b_gpu_Gk);
  g_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Gk, *b_gpu_Gk, *gpu_Rw_inv, *b_gpu_x);
  h_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ipiv);
  syncIsDirty = true;
  for (j = 0; j < 3; j++) {
    c = j * 5;
    ar = 2 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*b_gpu_x)[c]);
    for (ia = 0; ia <= ar; ia++) {
      ix++;
      s = std::abs((*b_gpu_x)[ix]);
      if (s > smax) {
        iy = ia + 1;
        smax = s;
      }
    }

    if ((*b_gpu_x)[c + iy] != 0.0) {
      if (iy != 0) {
        (*gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        ar = iy;
        for (ia = 0; ia < 4; ia++) {
          ix = j + ia * 4;
          iy = ar + ia * 4;
          smax = (*b_gpu_x)[ix];
          (*b_gpu_x)[ix] = (*b_gpu_x)[iy];
          (*b_gpu_x)[iy] = smax;
        }
      }

      i2 = (c - j) + 2;
      for (i = 0; i <= i2 - c; i++) {
        iy = (c + i) + 1;
        (*b_gpu_x)[iy] /= (*b_gpu_x)[c];
      }
    }

    ar = 2 - j;
    iy = c + 6;
    jy = c + 4;
    for (b_j = 0; b_j <= ar; b_j++) {
      smax = (*b_gpu_x)[jy];
      if ((*b_gpu_x)[jy] != 0.0) {
        ix = c;
        i2 = iy - 2;
        i3 = iy - j;
        for (kAcol = 0; kAcol <= i3 - i2; kAcol++) {
          i = (iy + kAcol) - 1;
          (*b_gpu_x)[i] += (*b_gpu_x)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 4;
      iy += 4;
    }
  }

  i_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_p);
  syncIsDirty = true;
  for (ia = 0; ia < 3; ia++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*gpu_ipiv)[ia] > ia + 1) {
      iy = (*gpu_p)[(*gpu_ipiv)[ia] - 1];
      (*gpu_p)[(*gpu_ipiv)[ia] - 1] = (*gpu_p)[ia];
      (*gpu_p)[ia] = static_cast<signed char>(iy);
    }
  }

  for (ia = 0; ia < 4; ia++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    i1 = (*gpu_p)[ia];
    (*gpu_Rw)[ia + (((*gpu_p)[ia] - 1) << 2)] = 1.0;
    for (j = 0; j <= 3 - ia; j++) {
      b_j = ia + j;
      if ((*gpu_Rw)[b_j + ((i1 - 1) << 2)] != 0.0) {
        for (i = 0; i <= 2 - b_j; i++) {
          iy = (b_j + i) + 1;
          (*gpu_Rw)[iy + ((i1 - 1) << 2)] -= (*gpu_Rw)[b_j + ((i1 - 1) << 2)] *
            (*b_gpu_x)[iy + (b_j << 2)];
        }
      }
    }
  }

  for (j = 0; j < 4; j++) {
    iy = (j << 2) - 1;
    for (ia = 0; ia < 4; ia++) {
      jy = 4 - ia;
      kAcol = (3 - ia) << 2;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Rw)[(iy - ia) + 4] != 0.0) {
        (*gpu_Rw)[(iy - ia) + 4] /= (*b_gpu_x)[(kAcol - ia) + 3];
        for (i = 0; i <= jy - 2; i++) {
          (*gpu_Rw)[(i + iy) + 1] -= (*gpu_Rw)[(iy - ia) + 4] * (*b_gpu_x)[i +
            kAcol];
        }
      }
    }
  }

  j_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Gk, *gpu_Ih, *b_gpu_Ih);
  k_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Rw, *b_gpu_Ih, *c_gpu_Ih);
  l_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*gpu_Gk, *c_gpu_Ih, *gpu_calcEq);

  // 4*4
  // 4*4
  hipDeviceSynchronize();
  (*gpu_ih)[0] = initialization_vec[2];
  (*gpu_ih)[1] = 0.0;
  (*gpu_ih)[2] = 0.0;
  (*gpu_ih)[3] = 0.0;
  (*gpu_ih)[4] = initialization_vec[2];
  m_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ih);
  n_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ih, *gpu_a, *gpu_ik);
  o_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ik, *gpu_Fk_inv, *gpu_ih);

  // 4*1
  p_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_calcEq, *gpu_ih, *gpu_ik);

  // 4*1
  q_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*gpu_Fk_inv);
  r_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>
    (*gpu_calcEq, *gpu_Ih, *gpu_x);
  s_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_ipiv);
  syncIsDirty = true;
  for (j = 0; j < 7; j++) {
    c = j * 9;
    kAcol = c - 4;
    ar = 6 - j;
    iy = 0;
    ix = c;
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    smax = std::abs((*gpu_x)[c]);
    for (ia = 0; ia <= ar; ia++) {
      ix++;
      s = std::abs((*gpu_x)[ix]);
      if (s > smax) {
        iy = ia + 1;
        smax = s;
      }
    }

    if ((*gpu_x)[c + iy] != 0.0) {
      if (iy != 0) {
        (*b_gpu_ipiv)[j] = static_cast<signed char>((j + iy) + 1);
        iy += j;
        ar = iy;
        for (ia = 0; ia < 8; ia++) {
          ix = j + ia * 8;
          iy = ar + ia * 8;
          smax = (*gpu_x)[ix];
          (*gpu_x)[ix] = (*gpu_x)[iy];
          (*gpu_x)[iy] = smax;
        }
      }

      i2 = (c - j) + 2;
      for (i = 0; i <= i2 - kAcol; i++) {
        iy = (c + i) + 1;
        (*gpu_x)[iy] /= (*gpu_x)[c];
      }
    }

    ar = 6 - j;
    iy = c + 10;
    jy = c + 8;
    for (b_j = 0; b_j <= ar; b_j++) {
      smax = (*gpu_x)[jy];
      if ((*gpu_x)[jy] != 0.0) {
        ix = c;
        i2 = iy - 6;
        i3 = iy - j;
        for (kAcol = 0; kAcol <= i3 - i2; kAcol++) {
          i = (iy + kAcol) - 1;
          (*gpu_x)[i] += (*gpu_x)[ix + 1] * -smax;
          ix++;
        }
      }

      jy += 8;
      iy += 8;
    }
  }

  t_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_p);
  syncIsDirty = true;
  for (ia = 0; ia < 7; ia++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    if ((*b_gpu_ipiv)[ia] > ia + 1) {
      iy = (*b_gpu_p)[(*b_gpu_ipiv)[ia] - 1];
      (*b_gpu_p)[(*b_gpu_ipiv)[ia] - 1] = (*b_gpu_p)[ia];
      (*b_gpu_p)[ia] = static_cast<signed char>(iy);
    }
  }

  for (ia = 0; ia < 8; ia++) {
    if (syncIsDirty) {
      hipDeviceSynchronize();
      syncIsDirty = false;
    }

    i1 = (*b_gpu_p)[ia];
    (*gpu_Fk_inv)[ia + (((*b_gpu_p)[ia] - 1) << 3)] = 1.0;
    for (j = 0; j <= 7 - ia; j++) {
      b_j = ia + j;
      if ((*gpu_Fk_inv)[b_j + ((i1 - 1) << 3)] != 0.0) {
        for (i = 0; i <= 6 - b_j; i++) {
          iy = (b_j + i) + 1;
          (*gpu_Fk_inv)[iy + ((i1 - 1) << 3)] -= (*gpu_Fk_inv)[b_j + ((i1 - 1) <<
            3)] * (*gpu_x)[iy + (b_j << 3)];
        }
      }
    }
  }

  for (j = 0; j < 8; j++) {
    iy = (j << 3) - 1;
    for (ia = 0; ia < 8; ia++) {
      jy = 8 - ia;
      kAcol = (7 - ia) << 3;
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      if ((*gpu_Fk_inv)[(iy - ia) + 8] != 0.0) {
        (*gpu_Fk_inv)[(iy - ia) + 8] /= (*gpu_x)[(kAcol - ia) + 7];
        for (i = 0; i <= jy - 2; i++) {
          (*gpu_Fk_inv)[(i + iy) + 1] -= (*gpu_Fk_inv)[(iy - ia) + 8] * (*gpu_x)
            [i + kAcol];
        }
      }
    }
  }

  // 4*4
  u_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ik, *gpu_Fk_inv, *gpu_ih);

  // 4*1
  // ---- Measurement step --------------------------
  //  Extract front and rear track widths
  //  drive axle 1
  //  drive axle 2
  smax = L_axlePos[0] + L_geometricWheelbase;

  // 1*1
  v_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_B_usedMeas_vec, *gpu_Re, *gpu_Re_inv);

  // 1*13
  w_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(192U, 1U, 1U)>>>
    (*b_gpu_Re_inv);
  x_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_Re_inv, *b_gpu_Re_inv);

  // 13*13
  //  Linear parts
  hipDeviceSynchronize();
  (*gpu_H_linear)[1] = 0.0;
  (*gpu_H_linear)[14] = 0.0;
  (*gpu_H_linear)[27] = 0.0;
  (*gpu_H_linear)[40] = L_imuToRear;
  (*gpu_H_linear)[53] = 0.0;
  (*gpu_H_linear)[66] = 0.0;
  (*gpu_H_linear)[79] = 0.0;
  (*gpu_H_linear)[92] = L_imuToRear;
  (*gpu_H_linear)[6] = 1.0;
  (*gpu_H_linear)[19] = 0.0;
  (*gpu_H_linear)[32] = -L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[45] = 0.0;
  (*gpu_H_linear)[58] = 1.0;
  (*gpu_H_linear)[71] = 0.0;
  (*gpu_H_linear)[84] = -L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[97] = 0.0;
  (*gpu_H_linear)[7] = 1.0;
  (*gpu_H_linear)[20] = 0.0;
  (*gpu_H_linear)[33] = L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[46] = 0.0;
  (*gpu_H_linear)[59] = 1.0;
  (*gpu_H_linear)[72] = 0.0;
  (*gpu_H_linear)[85] = L_trackWidth[2] / 2.0;
  (*gpu_H_linear)[98] = 0.0;
  (*gpu_H_linear)[8] = 1.0;
  (*gpu_H_linear)[21] = 0.0;
  (*gpu_H_linear)[34] = -L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[47] = 0.0;
  (*gpu_H_linear)[60] = 1.0;
  (*gpu_H_linear)[73] = 0.0;
  (*gpu_H_linear)[86] = -L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[99] = 0.0;
  (*gpu_H_linear)[9] = 1.0;
  (*gpu_H_linear)[22] = 0.0;
  (*gpu_H_linear)[35] = L_trackWidth[3] / 2.0;
  (*gpu_H_linear)[48] = 0.0;
  (*gpu_H_linear)[61] = 1.0;
  (*gpu_H_linear)[74] = 0.0;
  (*gpu_H_linear)[87] = L_trackWidth[3] / 2.0;
  y_InformationFilterUpdate_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_H_linear);
  ab_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_iv3, *gpu_iv4, *gpu_iv5, *gpu_H_linear);

  //  Nonlinear parts
  bb_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ih, *gpu_H_linear, *gpu_hk);

  // 13*1
  // the new hk 13*1 matrix is formed from this calculation
  hipDeviceSynchronize();
  (*gpu_hk)[4] = ((*gpu_ih)[0] - L_trackWidth[0] * (*gpu_ih)[2] / 2.0) * std::
    cos(delta) + smax * (*gpu_ih)[2] * std::sin(delta);
  (*gpu_hk)[5] = ((*gpu_ih)[0] + L_trackWidth[0] * (*gpu_ih)[2] / 2.0) * std::
    cos(delta) + smax * (*gpu_ih)[2] * std::sin(delta);
  (*gpu_H_linear)[30] = -L_trackWidth[0] * std::cos(delta) / 2.0 + smax * std::
    sin(delta);
  (*gpu_H_linear)[5] = std::cos(delta);
  (*gpu_H_linear)[31] = L_trackWidth[0] * std::cos(delta) / 2.0 + smax * std::
    sin(delta);
  cb_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(smax,
    delta, L_imuToRear, *gpu_ih, *gpu_H_linear, *gpu_hk);
  db_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(128U, 1U, 1U)>>>
    (*gpu_H_linear, *gpu_A);
  syncIsDirty = true;
  for (iy = 0; iy < 13; iy++) {
    kAcol = iy << 3;
    i2 = kAcol - 7;
    for (i = 0; i <= kAcol - i2; i++) {
      if (syncIsDirty) {
        hipDeviceSynchronize();
        syncIsDirty = false;
      }

      (*gpu_C)[kAcol + i] = 0.0;
    }
  }

  for (iy = 0; iy < 13; iy++) {
    jy = iy * 13 + 1;
    kAcol = iy << 3;
    ar = -1;
    i2 = jy - 12;
    for (b_j = 0; b_j <= jy - i2; b_j++) {
      ix = jy + b_j;
      ia = ar;
      i3 = kAcol - 6;
      c = kAcol + 1;
      for (i = 0; i <= c - i3; i++) {
        j = kAcol + i;
        ia++;
        if (syncIsDirty) {
          hipDeviceSynchronize();
          syncIsDirty = false;
        }

        (*gpu_C)[j] += (*b_gpu_Re_inv)[ix - 1] * (*gpu_A)[ia];
      }

      ar += 8;
    }
  }

  eb_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*gpu_ih, *gpu_H_linear, *gpu_hk, *gpu_y_meas, *b_gpu_y_meas);
  fb_InformationFilterUpdate_kern<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
    (*b_gpu_y_meas, *gpu_C, *gpu_ik, *gpu_op);

  // Hk is always a 13*4 matrix and hk is always a 13*1 matrix
  hipDeviceSynchronize();
  hipMemcpy(&op[0], gpu_op, 64UL, hipMemcpyDeviceToHost);
  hipFree(*gpu_y_meas);
  hipFree(*gpu_iv5);
  hipFree(*gpu_iv4);
  hipFree(*gpu_iv3);
  hipFree(*gpu_Re);
  hipFree(*gpu_B_usedMeas_vec);
  hipFree(*gpu_a);
  hipFree(*gpu_iv2);
  hipFree(*gpu_iv1);
  hipFree(*gpu_iv);
  hipFree(*b_gpu_Rw);
  hipFree(*gpu_op);
  hipFree(*gpu_Rw);
  hipFree(*gpu_Rw_inv);
  hipFree(*gpu_ipiv);
  hipFree(*gpu_p);
  hipFree(*gpu_Fk_inv);
  hipFree(*gpu_x);
  hipFree(*b_gpu_ipiv);
  hipFree(*b_gpu_p);
  hipFree(*gpu_Gk);
  hipFree(*gpu_Ih);
  hipFree(*b_gpu_Gk);
  hipFree(*b_gpu_x);
  hipFree(*b_gpu_Ih);
  hipFree(*c_gpu_Ih);
  hipFree(*gpu_calcEq);
  hipFree(*gpu_ih);
  hipFree(*gpu_ik);
  hipFree(*gpu_Re_inv);
  hipFree(*b_gpu_Re_inv);
  hipFree(*gpu_H_linear);
  hipFree(*gpu_hk);
  hipFree(*gpu_A);
  hipFree(*b_gpu_y_meas);
  hipFree(*gpu_C);
}

//
// Arguments    : void
// Return Type  : void
//
void InformationFilterUpdate_initialize()
{
}

//
// Arguments    : void
// Return Type  : void
//
void InformationFilterUpdate_terminate()
{
  // (no terminate code required)
}

//
// File trailer for InformationFilterUpdate.cu
//
// [EOF]
//
